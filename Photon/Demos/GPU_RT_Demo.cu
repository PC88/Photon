#include "hip/hip_runtime.h"
﻿#include "Demos\GPU_RT_Demo.h"
#include <hiprand/hiprand_kernel.h> // cuRAND
#include "ppm/ppm.hpp"

// test commit GPU

// credit: https://github.com/rogerallen/raytracinginoneweekendincuda/tree/master
// credit: https://developer.nvidia.com/blog/accelerated-ray-tracing-cuda/
// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

// credit: https://github.com/rogerallen/raytracinginoneweekendincuda/tree/master
// credit: https://developer.nvidia.com/blog/accelerated-ray-tracing-cuda/
// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
	if (result)
	{
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

__global__ void render(vec3* fb, int max_x, int max_y)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	fb[pixel_index] = vec3(float(i) / max_x, float(j) / max_y, 0.2f);
}

GPU_RT_Demo::GPU_RT_Demo()
{
	int nx = 1200;
	int ny = 600;
	int tx = 8;
	int ty = 8;

	std::cerr << "Rendering a " << nx << "x" << ny << " image ";
	std::cerr << "in " << tx << "x" << ty << " blocks.\n";

	int num_pixels = nx * ny;
	size_t fb_size = num_pixels * sizeof(vec3);

	// allocate FB
	vec3* fb;
	checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

	clock_t start, stop;
	start = clock();
	// Render our buffer
	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);
	render<<<blocks, threads>>>(fb, nx, ny);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	stop = clock();
	double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
	std::cerr << "took " << timer_seconds << " seconds.\n";

	// Output FB as Image
	std::cout << "P3\n" << nx << " " << ny << "\n255\n";
	for (int j = ny - 1; j >= 0; j--) 
	{
		for (int i = 0; i < nx; i++) 
		{
			size_t pixel_index = j * nx + i;
			int ir = int(255.99 * fb[pixel_index].r());
			int ig = int(255.99 * fb[pixel_index].g());
			int ib = int(255.99 * fb[pixel_index].b());
			std::cout << ir << " " << ig << " " << ib << "\n";
		}
	}

	checkCudaErrors(hipFree(fb));
}

GPU_RT_Demo::~GPU_RT_Demo()
{

}

